#include "hip/hip_runtime.h"
#include "util.h"
#include "kernel.h"
#include "hip/hip_runtime.h"
#include "math.h"

__constant__ FluxKernelArgs flux_ctx;
__constant__ DtKernelArgs dt_ctx;
__constant__ RKKernelArgs rk_ctx;

void init_allocate(){
	for (int i=0; i<3; i++){
		hipHostAlloc(&BCArgs[i], sizeof(collBCKernelArgs), hipHostMallocWriteCombined);
		//hipHostMalloc(&BCArgs[i], sizeof(collBCKernelArgs));

		hipHostAlloc(&fluxArgs[i], sizeof(FluxKernelArgs), hipHostMallocWriteCombined);

		hipHostAlloc(&RKArgs[i], sizeof(RKKernelArgs), hipHostMallocWriteCombined);

		hipHostAlloc(&dtArgs, sizeof(DtKernelArgs), hipHostMallocWriteCombined);

		hipHostAlloc(&dt_host, sizeof(float), hipHostMallocWriteCombined);
		//hipHostMalloc(&fluxArgs[i], sizeof(FluxKernelArgs));
	}
}

__global__ void RKKernel(int step){ 

	float dt = rk_ctx.dt[0];
	float u0,u1,u2,u3,r0,r1,r2,r3,q0,q1,q2,q3;
	int global_border = rk_ctx.global_border;

	// Global indexes     
        int xid = blockIdx.x*blockDim.x + threadIdx.x - global_border;
        int yid = blockIdx.y*blockDim.y + threadIdx.y - global_border;

        if ( xid < 0 || xid >= rk_ctx.nx || yid < 0 || yid >= rk_ctx.ny ) return; 

	u0 = global_index(rk_ctx.U0.ptr, rk_ctx.U0.pitch, xid, yid, global_border)[0];
	u1 = global_index(rk_ctx.U1.ptr, rk_ctx.U1.pitch, xid, yid, global_border)[0];
	u2 = global_index(rk_ctx.U2.ptr, rk_ctx.U2.pitch, xid, yid, global_border)[0];
	u3 = global_index(rk_ctx.U3.ptr, rk_ctx.U3.pitch, xid, yid, global_border)[0];		

	r0 = global_index(rk_ctx.R0.ptr, rk_ctx.R0.pitch, xid, yid, global_border)[0];
        r1 = global_index(rk_ctx.R1.ptr, rk_ctx.R1.pitch, xid, yid, global_border)[0];
        r2 = global_index(rk_ctx.R2.ptr, rk_ctx.R2.pitch, xid, yid, global_border)[0];
        r3 = global_index(rk_ctx.R3.ptr, rk_ctx.R3.pitch, xid, yid, global_border)[0];

	if (step == 0) {
		q0 =  u0 + dt*r0;
		q1 =  u1 + dt*r1;
		q2 =  u2 + dt*r2;
		q3 =  u3 + dt*r3;
	}
	else {
		q0 = global_index(rk_ctx.Q0.ptr, rk_ctx.Q0.pitch, xid, yid, global_border)[0];
        	q1 = global_index(rk_ctx.Q1.ptr, rk_ctx.Q1.pitch, xid, yid, global_border)[0];
        	q2 = global_index(rk_ctx.Q2.ptr, rk_ctx.Q2.pitch, xid, yid, global_border)[0];
        	q3 = global_index(rk_ctx.Q3.ptr, rk_ctx.Q3.pitch, xid, yid, global_border)[0];

		q0 = 0.5f*(q0 + (u0 + dt*r0));
		q1 = 0.5f*(q1 + (u1 + dt*r1));
		q2 = 0.5f*(q2 + (u2 + dt*r2));
		q3 = 0.5f*(q3 + (u3 + dt*r3));
	}

	global_index(rk_ctx.Q0.ptr, rk_ctx.Q0.pitch, xid, yid, global_border)[0] = q0;
        global_index(rk_ctx.Q1.ptr, rk_ctx.Q1.pitch, xid, yid, global_border)[0] = q1;
        global_index(rk_ctx.Q2.ptr, rk_ctx.Q2.pitch, xid, yid, global_border)[0] = q2;
        global_index(rk_ctx.Q3.ptr, rk_ctx.Q3.pitch, xid, yid, global_border)[0] = q3;
}

void callRKKernel(dim3 grid, dim3 block, int step, RKKernelArgs* h_ctx){	
	hipMemcpyToSymbolAsync(HIP_SYMBOL(rk_ctx), h_ctx, sizeof(RKKernelArgs), 0, hipMemcpyHostToDevice);
	RKKernel<<<grid, block>>>(step);
}

__global__ void DtKernel(int nThreads){

	extern __shared__ float sdata[];
	volatile float* sdata_volatile = sdata;
	unsigned int tid = threadIdx.x;
	int threads = nThreads;
	float dt;
	//printf("THREADID %i",tid);

	sdata[tid] = FLT_MAX;

	for (unsigned int i=tid; i<dt_ctx.nElements; i += threads)
		sdata[tid] = min(sdata[tid], dt_ctx.L[i]);
		__syncthreads();
	//	if (tid == 0){
	//		printf("START\n");
	//		for (int k=0; k<nThreads; k++)
	//			printf(" %.5f\t",sdata[k]);  
	//	}	
	//Now, reduce all elements into a single element
	if (threads >= 512) {
		if (tid < 256) sdata[tid] = min(sdata[tid], sdata[tid + 256]);
		__syncthreads();
	}
	if (threads >= 256) {
		if (tid < 128) sdata[tid] = min(sdata[tid], sdata[tid + 128]);
		__syncthreads();
	}
	if (threads >= 128) {
		if (tid < 64) sdata[tid] = min(sdata[tid], sdata[tid + 64]);
		__syncthreads();
	}
	if (tid < 32) {
		if (threads >= 64) sdata_volatile[tid] = min(sdata_volatile[tid], sdata_volatile[tid + 32]);
		if (tid < 16) {
			if (threads >= 32) sdata_volatile[tid] = min(sdata_volatile[tid], sdata_volatile[tid + 16]);
			if (threads >= 16) sdata_volatile[tid] = min(sdata_volatile[tid], sdata_volatile[tid +  8]);
			if (threads >=  8) sdata_volatile[tid] = min(sdata_volatile[tid], sdata_volatile[tid +  4]);
			if (threads >=  4) sdata_volatile[tid] = min(sdata_volatile[tid], sdata_volatile[tid +  2]);
			if (threads >=  2) sdata_volatile[tid] = min(sdata_volatile[tid], sdata_volatile[tid +  1]);
		}

		if (tid == 0) {
			dt = sdata_volatile[tid];
			if (dt == FLT_MAX) {
				//If no water at all, and no sources, 
				//we really do not need to simulate, 
				//but using FLT_MAX will make things crash...
				dt = 1.0e-7f;
			}
			dt_ctx.dt[tid] = dt*dt_ctx.scale;
		//	printf("TID %i",tid); 
		}
	}
}


void callDtKernel(int nThreads, DtKernelArgs* h_ctx){

	hipMemcpyToSymbolAsync(HIP_SYMBOL(dt_ctx), h_ctx, sizeof(DtKernelArgs), 0, hipMemcpyHostToDevice);	
	DtKernel<<<1,nThreads,sizeof(float)*nThreads>>>(nThreads);
}

inline __device__ void fluxAndLambdaFuncF(float& rho, float& U1, float& U2, float& U3,
		const float& gamma,
		float& F0, float& F1, float& F2, float& F3,
		float& u, float& v,float& c){

	float pressure, E;

	// Vaues needed to compute the eigenvalues
	u = U1/rho;
	v = U2/rho;
	E = U3;
        pressure = (gamma - 1.0f)*(E-0.5f*rho*(u*u + v*v));
        c = sqrtf(gamma*pressure/rho);	      

	// Flux computation
	F0 = U1;
	F1 = U1*u +  pressure;
	F2 = U1*v;
	F3 = u*(E+pressure);
}

inline __device__ void fluxAndLambdaFuncG(float& rho, float& U1, float& U2, float& U3,
                const float& gamma,
                float& G0, float& G1, float& G2, float& G3,
                float& u, float& v,float& c){
        
        float pressure, E;

	// Vaues needed to compute the eigenvalues
	u = U1/rho;
	v = U2/rho;
	E = U3;
        pressure = (gamma - 1.0f)*(E-0.5f*rho*(u*u + v*v));
	c = sqrtf(gamma*pressure/rho);
        //if (pressure < 0)
                //printf("ZERO alert compute G and Lambda gamma:%.3f pressure: %.3f rho:%.3f  rho_u:%.3f rho_v%.3f E%.3f\n", gamma,pressure,rho,U1,U2,E);
        // Flux computation
        G0 = U2;
        G1 = U2*u;
        G2 = U2*v + pressure;
        G3 = v*(E+pressure);
}

inline __device__ float minEigenVal(float a, float b) {
	return fminf(fminf(a, b), 0.0f);
}

inline __device__ float maxEigenVal(float a, float b) {
	return fmaxf(fmaxf(a, b), 0.0f);
}

inline __device__ float sign(float& a) {
	/**
	  * The following works by bit hacks. In non-obfuscated code, something like
	  *  float r = ((int&)a & 0x7FFFFFFF)!=0; //set r to one or zero
	  *  (int&)r |= ((int&)a & 0x80000000);   //Copy sign bit of a
	  *  return r;
	  */
#ifndef NEW_SIGN
	return (signed((int&)a & 0x80000000) >> 31 ) | ((int&)a & 0x7FFFFFFF)!=0;
#else
	float r = ((int&)a & 0x7FFFFFFF)!=0;
	return copysignf(r, a);
#endif
}


inline __device__ float minmod(float a, float b, float c){
	return 0.25f
	*sign(a)
	*(sign(a) + sign(b))
	*(sign(b) + sign(c))
	*fminf( fminf(fabsf(a), fabsf(b)), fabsf(c) );


/*	if ( a > 0 && b > 0 && c > 0)
		return fminf(c,fminf(a,b));
	else if ( a < 0 && b < 0 && c < 0)
		return fmaxf(c,fmaxf(a,b));
	else
		return 0.0;
*/
}

inline __device__ float limiter(float u_plus, float u_center, float u_minus){
	return minmod(flux_ctx.theta*(u_plus-u_center),(u_plus-u_minus)*0.5f, flux_ctx.theta*(u_center-u_minus));
}

inline __device__ void reconstructPointVal(float (&U)[4][BLOCKDIM_X][SM_BLOCKDIM_Y], float (&Ux)[4][BLOCKDIM_X][SM_BLOCKDIM_Y], float (&Uy)[4][BLOCKDIM_X][SM_BLOCKDIM_Y], unsigned int i, unsigned int j){
	float u_center,u_south,u_north,u_east,u_west;

	float ux_out, uy_out;

	for (int l=0; l<4; l++){
			u_center = U[l][i][j];
                        u_south = U[l][i][j-1];
                        u_north = U[l][i][j+1];
                        u_west = U[l][i-1][j];
                        u_east = U[l][i+1][j];

			// Compute interface values, each cell computes 
                        ux_out = 0.5f*limiter(u_east, u_center, u_west);
                        uy_out = 0.5f*limiter(u_north, u_center, u_south);

			Ux[l][i][j] = ux_out;
			Uy[l][i][j] = uy_out;			
	}

}

inline __device__ float computeFluxWest(float (&U)[4][BLOCKDIM_X][SM_BLOCKDIM_Y], float (&Ux)[4][BLOCKDIM_X][SM_BLOCKDIM_Y], float (&Uy)[4][BLOCKDIM_X][SM_BLOCKDIM_Y], float (&Uout)[4][BLOCKDIM_X][SM_BLOCKDIM_Y], unsigned int i, unsigned int j){
	
	float U0ma, U1ma, U2ma, U3ma;
	float U0mb, U1mb, U2mb, U3mb;
	float U0pa, U1pa, U2pa, U3pa;
	float U0pb, U1pb, U2pb, U3pb;
	float FG0pa, FG1pa, FG2pa, FG3pa;
	float FG0pb, FG1pb, FG2pb, FG3pb;
	float FG0ma, FG1ma, FG2ma, FG3ma;
	float FG0mb, FG1mb, FG2mb, FG3mb;
	float upa,vpa,cpa,uma,vma,cma;
	float upb,vpb,cpb,umb,vmb,cmb;
	float ama, apa;
	float amb, apb;
	float alpha = 1.0/sqrtf(3);
	
	// The eastern reconstruction point of u(i-1,j)
	U0ma = U[0][i-1][j] + Ux[0][i-1][j] + 2*alpha*Uy[0][i-1][j];
	U1ma = U[1][i-1][j] + Ux[1][i-1][j] + 2*alpha*Uy[1][i-1][j];
	U2ma = U[2][i-1][j] + Ux[2][i-1][j] + 2*alpha*Uy[2][i-1][j];
	U3ma = U[3][i-1][j] + Ux[3][i-1][j] + 2*alpha*Uy[3][i-1][j];
	
	// The eastern reconstruction point of u(i-1,j)
	U0mb = U[0][i-1][j] + Ux[0][i-1][j] - 2*alpha*Uy[0][i-1][j];
	U1mb = U[1][i-1][j] + Ux[1][i-1][j] - 2*alpha*Uy[1][i-1][j];
	U2mb = U[2][i-1][j] + Ux[2][i-1][j] - 2*alpha*Uy[2][i-1][j];
	U3mb = U[3][i-1][j] + Ux[3][i-1][j] - 2*alpha*Uy[3][i-1][j];
	
	// The western reconstruction point of u(i,j)
	U0pa = U[0][i][j] - Ux[0][i][j] + 2*alpha*Uy[0][i][j];
        U1pa = U[1][i][j] - Ux[1][i][j] + 2*alpha*Uy[1][i][j];
        U2pa = U[2][i][j] - Ux[2][i][j] + 2*alpha*Uy[2][i][j];
        U3pa = U[3][i][j] - Ux[3][i][j] + 2*alpha*Uy[3][i][j];
		
	// The western reconstruction point of u(i,j)
	U0pb = U[0][i][j] - Ux[0][i][j] - 2*alpha*Uy[0][i][j];
        U1pb = U[1][i][j] - Ux[1][i][j] - 2*alpha*Uy[1][i][j];
        U2pb = U[2][i][j] - Ux[2][i][j] - 2*alpha*Uy[2][i][j];
        U3pb = U[3][i][j] - Ux[3][i][j] - 2*alpha*Uy[3][i][j];

	fluxAndLambdaFuncF(U0pa, U1pa, U2pa, U3pa, flux_ctx.gamma, FG0pa, FG1pa, FG2pa, FG3pa, upa, vpa, cpa);
	fluxAndLambdaFuncF(U0ma, U1ma, U2ma, U3ma, flux_ctx.gamma, FG0ma, FG1ma, FG2ma, FG3ma, uma, vma, cma);

	fluxAndLambdaFuncF(U0pb, U1pb, U2pb, U3pb, flux_ctx.gamma, FG0pb, FG1pb, FG2pb, FG3pb, upb, vpb, cpb);
	fluxAndLambdaFuncF(U0mb, U1mb, U2mb, U3mb, flux_ctx.gamma, FG0mb, FG1mb, FG2mb, FG3mb, umb, vmb, cmb);

	ama = minEigenVal(uma-cma, upa-cpa);
	apa = maxEigenVal(uma+cma, upa+cpa);

	amb = minEigenVal(umb-cmb, upb-cpb);
	apb = maxEigenVal(umb+cmb, upb+cpb);

	__syncthreads();
	
	float flux0a = ((apa*FG0ma - ama*FG0pa) + apa*ama*(U0pa-U0ma))/(apa-ama);
	float flux1a = ((apa*FG1ma -ama*FG1pa) + apa*ama*(U1pa-U1ma))/(apa-ama);
	float flux2a = ((apa*FG2ma -ama*FG2pa) + apa*ama*(U2pa-U2ma))/(apa-ama);
	float flux3a = ((apa*FG3ma -ama*FG3pa) + apa*ama*(U3pa-U3ma))/(apa-ama);

	float flux0b = ((apb*FG0mb -amb*FG0pb) + apb*amb*(U0pb-U0mb))/(apb-amb);
	float flux1b = ((apb*FG1mb -amb*FG1pb) + apb*amb*(U1pb-U1mb))/(apb-amb);
	float flux2b = ((apb*FG2mb -amb*FG2pb) + apb*amb*(U2pb-U2mb))/(apb-amb);
	float flux3b = ((apb*FG3mb -amb*FG3pb) + apb*amb*(U3pb-U3mb))/(apb-amb);
	
	Uout[0][i][j] = 0.5f*(flux0a + flux0b);
	Uout[1][i][j] = 0.5f*(flux1a + flux1b);
	Uout[2][i][j] = 0.5f*(flux2a + flux2b);
	Uout[3][i][j] = 0.5f*(flux3a + flux3b);	

	apa = 0.5*(apa+apb);
	ama = 0.5*(ama+amb);	

	return flux_ctx.dx/fmaxf(apa, -ama);
}
 	
inline __device__ float computeFluxSouth(float (&U)[4][BLOCKDIM_X][SM_BLOCKDIM_Y], float (&Ux)[4][BLOCKDIM_X][SM_BLOCKDIM_Y], float (&Uy)[4][BLOCKDIM_X][SM_BLOCKDIM_Y], float (&Uout)[4][BLOCKDIM_X][SM_BLOCKDIM_Y], unsigned int i, unsigned int j){

	float U0ma, U1ma, U2ma, U3ma;
	float U0mb, U1mb, U2mb, U3mb;
	float U0pa, U1pa, U2pa, U3pa;
	float U0pb, U1pb, U2pb, U3pb;
	float FG0pa, FG1pa, FG2pa, FG3pa;
	float FG0pb, FG1pb, FG2pb, FG3pb;
	float FG0ma, FG1ma, FG2ma, FG3ma;
	float FG0mb, FG1mb, FG2mb, FG3mb;
	float upa,vpa,cpa,uma,vma,cma;
	float upb,vpb,cpb,umb,vmb,cmb;
	float ama, apa;
	float amb, apb;
	float alpha = 1.0/sqrtf(3);

        // The eastern reconstruction point of u(i-1,j)
        U0ma = U[0][i][j-1] + Uy[0][i][j-1] + 2*alpha*Ux[0][i][j-1];
        U1ma = U[1][i][j-1] + Uy[1][i][j-1] + 2*alpha*Ux[1][i][j-1];
        U2ma = U[2][i][j-1] + Uy[2][i][j-1] + 2*alpha*Ux[2][i][j-1];
        U3ma = U[3][i][j-1] + Uy[3][i][j-1] + 2*alpha*Ux[3][i][j-1];
 
       // The eastern reconstruction point of u(i-1,j)
        U0mb = U[0][i][j-1] + Uy[0][i][j-1] - 2*alpha*Ux[0][i][j-1];
        U1mb = U[1][i][j-1] + Uy[1][i][j-1] - 2*alpha*Ux[1][i][j-1];
        U2mb = U[2][i][j-1] + Uy[2][i][j-1] - 2*alpha*Ux[2][i][j-1];
        U3mb = U[3][i][j-1] + Uy[3][i][j-1] - 2*alpha*Ux[3][i][j-1];

        // The western reconstruction point of u(i,j)
        U0pa = U[0][i][j] - Uy[0][i][j] + 2*alpha*Ux[0][i][j];
        U1pa = U[1][i][j] - Uy[1][i][j] + 2*alpha*Ux[1][i][j];
        U2pa = U[2][i][j] - Uy[2][i][j] + 2*alpha*Ux[2][i][j];
        U3pa = U[3][i][j] - Uy[3][i][j] + 2*alpha*Ux[3][i][j];

        // The western reconstruction point of u(i,j)
        U0pb = U[0][i][j] - Uy[0][i][j] - 2*alpha*Ux[0][i][j];
        U1pb = U[1][i][j] - Uy[1][i][j] - 2*alpha*Ux[1][i][j];
        U2pb = U[2][i][j] - Uy[2][i][j] - 2*alpha*Ux[2][i][j];
        U3pb = U[3][i][j] - Uy[3][i][j] - 2*alpha*Ux[3][i][j];

	fluxAndLambdaFuncG(U0pa, U1pa, U2pa, U3pa, flux_ctx.gamma, FG0pa, FG1pa, FG2pa, FG3pa, upa, vpa, cpa);
	fluxAndLambdaFuncG(U0ma, U1ma, U2ma, U3ma, flux_ctx.gamma, FG0ma, FG1ma, FG2ma, FG3ma, uma, vma, cma);

	fluxAndLambdaFuncG(U0pb, U1pb, U2pb, U3pb, flux_ctx.gamma, FG0pb, FG1pb, FG2pb, FG3pb, upb, vpb, cpb);
	fluxAndLambdaFuncG(U0mb, U1mb, U2mb, U3mb, flux_ctx.gamma, FG0mb, FG1mb, FG2mb, FG3mb, umb, vmb, cmb);

	ama = minEigenVal(vma-cma, vpa-cpa);
	apa = maxEigenVal(vma+cma, vpa+cpa);

	amb = minEigenVal(vmb-cmb, vpb-cpb);
	apb = maxEigenVal(vmb+cmb, vpb+cpb);

	__syncthreads();
	
	float flux0a = ((apa*FG0ma -ama*FG0pa) + apa*ama*(U0pa-U0ma))/(apa-ama);
	float flux1a = ((apa*FG1ma -ama*FG1pa) + apa*ama*(U1pa-U1ma))/(apa-ama);
	float flux2a = ((apa*FG2ma -ama*FG2pa) + apa*ama*(U2pa-U2ma))/(apa-ama);
	float flux3a = ((apa*FG3ma -ama*FG3pa) + apa*ama*(U3pa-U3ma))/(apa-ama);

	float flux0b = ((apb*FG0mb -amb*FG0pb) + apb*amb*(U0pb-U0mb))/(apb-amb);
	float flux1b = ((apb*FG1mb -amb*FG1pb) + apb*amb*(U1pb-U1mb))/(apb-amb);
	float flux2b = ((apb*FG2mb -amb*FG2pb) + apb*amb*(U2pb-U2mb))/(apb-amb);
	float flux3b = ((apb*FG3mb -amb*FG3pb) + apb*amb*(U3pb-U3mb))/(apb-amb);
	
	Uout[0][i][j] = 0.5f*(flux0a + flux0b);
	Uout[1][i][j] = 0.5f*(flux1a + flux1b);
	Uout[2][i][j] = 0.5f*(flux2a + flux2b);
	Uout[3][i][j] = 0.5f*(flux3a + flux3b);	

	apa = 0.5*(apa+apb);
	ama = 0.5*(ama+amb);	

	return flux_ctx.dx/fmaxf(apa, -ama);
}


__global__ void fluxKernel(int step){

	int global_border = flux_ctx.global_border;
	float dx = flux_ctx.dx;
	float dy = flux_ctx.dy;
 
	// Global indexes, multiply by tiledim because each block has a halo/border	
	int xid = blockIdx.x*flux_ctx.innerDimX + threadIdx.x - global_border;
	int yid = blockIdx.y*flux_ctx.innerDimY + threadIdx.y - global_border;

	xid = fminf(xid, flux_ctx.nx+global_border-1);
	yid = fminf(yid, flux_ctx.ny+global_border-1);

	// Local id
	int i = threadIdx.x;
	int j = threadIdx.y;

	float r = FLT_MAX;
	float r0, r1, r2, r3;

	const int nthreads = BLOCKDIM_X*BLOCKDIM_Y;

	__shared__ float timeStep[BLOCKDIM_X][BLOCKDIM_Y];
	timeStep[i][j] = FLT_MAX;

	__shared__ float local_U[4][BLOCKDIM_X][SM_BLOCKDIM_Y];
	__shared__ float local_Ux[4][BLOCKDIM_X][SM_BLOCKDIM_Y];
	__shared__ float local_Uy[4][BLOCKDIM_X][SM_BLOCKDIM_Y];
	__shared__ float local_Uoutx[4][BLOCKDIM_X][SM_BLOCKDIM_Y];
	__shared__ float local_Uouty[4][BLOCKDIM_X][SM_BLOCKDIM_Y];

	local_U[0][i][j] = global_index(flux_ctx.U0.ptr, flux_ctx.U0.pitch, xid, yid, global_border)[0];
	local_U[1][i][j] = global_index(flux_ctx.U1.ptr, flux_ctx.U1.pitch, xid, yid, global_border)[0];
	local_U[2][i][j] = global_index(flux_ctx.U2.ptr, flux_ctx.U2.pitch, xid, yid, global_border)[0];
	local_U[3][i][j] = global_index(flux_ctx.U3.ptr, flux_ctx.U3.pitch, xid, yid, global_border)[0];	

	__syncthreads();

	if ( i > 0 && i < BLOCKDIM_X - 1 && j > 0 && j < BLOCKDIM_Y - 1){
		reconstructPointVal(local_U, local_Ux, local_Uy, i, j);
	}

	__syncthreads();


	if ( i > 1 && i < TILEDIM_X + 1 && j > 1 && j < TILEDIM_Y)
		r = min(r, computeFluxWest(local_U, local_Ux, local_Uy, local_Uoutx,i, j));
	if ( i > 1 && i < TILEDIM_X  && j > 1 && j < TILEDIM_Y + 1)
		r = computeFluxSouth(local_U, local_Ux, local_Uy, local_Uouty, i, j);

	int p = threadIdx.y*blockDim.x+threadIdx.x;

	__syncthreads();

	if (xid > -1 && xid < flux_ctx.nx && yid > -1 && yid < flux_ctx.ny){
		if ( i > 1 && i < TILEDIM_X  && j > 1 && j < TILEDIM_Y){

			r0 = (local_Uoutx[0][i][j] - local_Uoutx[0][i+1][j])/dx + (local_Uouty[0][i][j] - local_Uouty[0][i][j+1])/dy;	
			r1 = (local_Uoutx[1][i][j] - local_Uoutx[1][i+1][j])/dx + (local_Uouty[1][i][j] - local_Uouty[1][i][j+1])/dy;   
			r2 = (local_Uoutx[2][i][j] - local_Uoutx[2][i+1][j])/dx + (local_Uouty[2][i][j] - local_Uouty[2][i][j+1])/dy;   
			r3 = (local_Uoutx[3][i][j] - local_Uoutx[3][i+1][j])/dx + (local_Uouty[3][i][j] - local_Uouty[3][i][j+1])/dy;   

			global_index(flux_ctx.R0.ptr, flux_ctx.R0.pitch, xid, yid, global_border)[0] = r0;//local_Ux[0][i][j]; 		
			global_index(flux_ctx.R1.ptr, flux_ctx.R1.pitch, xid, yid, global_border)[0] = r1;
			global_index(flux_ctx.R2.ptr, flux_ctx.R2.pitch, xid, yid, global_border)[0] = r2;
			global_index(flux_ctx.R3.ptr, flux_ctx.R3.pitch, xid, yid, global_border)[0] = r3;//local_Uy[0][i][j];

			timeStep[0][p] = r;

		}
	}

//Now, find and write out the maximal eigenvalue in this block
	if (step==0) {
	//	__syncthreads();
		volatile float* B_volatile = timeStep[0];
		//int p = threadIdx.y*blockDim.x+threadIdx.x; //reuse p for indexing
		//printf(" %i ", p);
		//Write the maximum eigenvalues computed by this thread into shared memory
		//Only consider eigenvalues within the internal domain
	/*	if (xid < flux_ctx.nx && yid < flux_ctx.ny && xid >= 0 && yid >=0){
			timeStep[0][p] = r; 
		}	
	*/
		__syncthreads();		

		//First use all threads to reduce min(1024, nthreads) values into 64 values
		//This first outer test is a compile-time test simply to remove statements if nthreads is less than 512.
		if (nthreads >= 512) {
			//This inner test (p < 512) first checks that the current thread should
			//be active in the reduction from min(1024, nthreads) elements to 512. Makes little sense here, but
			//a lot of sense for the last test where there should only be 64 active threads.
			//The second part of this test ((p+512) < nthreads) removes the threads that would generate an
			//out-of-bounds access to shared memory
			if (p < 512 && (p+512) < nthreads) timeStep[0][p] = fminf(timeStep[0][p], timeStep[0][p + 512]); //min(1024, nthreads)=>512
			__syncthreads();
		}

		if (nthreads >= 256) { 
			if (p < 256 && (p+256) < nthreads) timeStep[0][p] = fminf(timeStep[0][p], timeStep[0][p + 256]); //min(512, nthreads)=>256
			__syncthreads();
		}
		if (nthreads >= 128) {
			if (p < 128 && (p+128) < nthreads) timeStep[0][p] = fminf(timeStep[0][p], timeStep[0][p + 128]); //min(256, nthreads)=>128
			__syncthreads();
		}
		if (nthreads >= 64) {
			if (p < 64 && (p+64) < nthreads) timeStep[0][p] = fminf(timeStep[0][p], timeStep[0][p + 64]); //min(128, nthreads)=>64
			__syncthreads();
		}

		//Let the last warp reduce 64 values into a single value
		//Will generate out-of-bounds errors for nthreads < 64
		if (p < 32) {
			if (nthreads >= 64) B_volatile[p] = fminf(B_volatile[p], B_volatile[p + 32]); //64=>32
			if (nthreads >= 32) B_volatile[p] = fminf(B_volatile[p], B_volatile[p + 16]); //32=>16
			if (nthreads >= 16) B_volatile[p] = fminf(B_volatile[p], B_volatile[p +  8]); //16=>8
			if (nthreads >=  8) B_volatile[p] = fminf(B_volatile[p], B_volatile[p +  4]); //8=>4
			if (nthreads >=  4) B_volatile[p] = fminf(B_volatile[p], B_volatile[p +  2]); //4=>2
			if (nthreads >=  2) B_volatile[p] = fminf(B_volatile[p], B_volatile[p +  1]); //2=>1
		}

		if (threadIdx.y + threadIdx.x == 0) flux_ctx.L[blockIdx.x*gridDim.y + blockIdx.y] = B_volatile[0];


	}

}

void callFluxKernel(dim3 grid, dim3 block, int step, FluxKernelArgs* h_ctx){

	hipMemcpyToSymbolAsync(HIP_SYMBOL(flux_ctx), h_ctx, sizeof(FluxKernelArgs), 0, hipMemcpyHostToDevice);
	fluxKernel<<<grid, block>>>(step);
}



// Set wall boundry condition
__global__ void setBCPeriodic(gpu_raw_ptr U, unsigned int NX, unsigned int NY, int border){

	int threads = blockDim.x*blockDim.y;	

	float* B_in;
	float* B_out;

	int nx = NX-2*border;
	int ny = NY-2*border;

	int tid = threadIdx.y*blockDim.x+threadIdx.x;

	int kin;
	int kk;

	// SOUTH
	for (int b = 0; b < border; b++){
		B_out = global_index(U.ptr, U.pitch, 0, -1 - b, border);   
		B_in = global_index(U.ptr, U.pitch, 0, ny -1 - b, border);
		for (int k = tid; k < nx+border*2; k+=threads){
			kk = k-border;
			kin = min(kk,nx-1);
			kin = max(kin,0);			
			B_out[kk] = B_in[kin];
		}
	}

	// NORTH
	for (int b = 0; b < border; b++){
                B_out = global_index(U.ptr, U.pitch, 0, ny + b, border);   
                B_in = global_index(U.ptr, U.pitch, 0, 0 + b, border);
		for (int k = tid; k < nx+border*2; k+=threads){
			kk = k-border;
			kin = min(kk,nx-1);
			kin = max(kin,0);			
			B_out[kk] = B_in[kin];
		}

        }

	// WEST
	for (int k = tid; k < ny+border*2; k+= threads){
		kk = k-border;
        	B_out = global_index(U.ptr, U.pitch, 0, kk, border); 	
		kin = min(kk,ny-1);
		kin = max(kin,0);			
		for (int b = 0; b < border; b++)
                	B_out[-1-b] = global_index(U.ptr, U.pitch, nx -1 - b, kin, border)[0];                      
        }

	// EAST
        for (int k = tid; k < ny+border*2; k+= threads){
		kk = k-border;
                B_out = global_index(U.ptr, U.pitch, nx, kk, border);     
		kin = min(kk,ny-1);
		kin = max(kin,0);			
                for (int b = 0; b < border; b++)
                        B_out[b] = global_index(U.ptr, U.pitch, 0 + b, kin,border)[0];
        }
}


void callSetBCPeriodic(dim3 grid, dim3 block, gpu_raw_ptr U, unsigned int NX, unsigned int NY, int border){
	setBCPeriodic<<<grid, block>>>(U, NX, NY, border);
}	 

void callCollectiveSetBCPeriodic(dim3 grid, dim3 block, const collBCKernelArgs* arg){

	callSetBCPeriodic(grid, block, arg->U0, arg->NX, arg->NY, arg->global_border); 
	callSetBCPeriodic(grid, block, arg->U1, arg->NX, arg->NY, arg->global_border);
        callSetBCPeriodic(grid, block, arg->U2, arg->NX, arg->NY, arg->global_border);
        callSetBCPeriodic(grid, block, arg->U3, arg->NX, arg->NY, arg->global_border);
}


// Set wall boundry condition
__global__ void setBCOpen(gpu_raw_ptr U, unsigned int NX, unsigned int NY, int border){

	int threads = blockDim.x*blockDim.y;	

	float* B_in;
	float* B_out;

	int nx = NX-2*border;
	int ny = NY-2*border;

	int tid = threadIdx.y*blockDim.x+threadIdx.x;

	int kin;
	int kk;

	// SOUTH
	for (int b = 0; b < border; b++){
		B_out = global_index(U.ptr, U.pitch, 0, -1 - b, border);   
		B_in = global_index(U.ptr, U.pitch, 0, 0, border);
		for (int k = tid; k < nx+border*2; k+=threads){
			kk = k-border;
			kin = min(kk,nx-1);
			kin = max(kin,0);			
			B_out[kk] = B_in[kin];
		}
	}
	// NORTH
	for (int b = 0; b < border; b++){
                B_out = global_index(U.ptr, U.pitch, 0, ny + b, border);   
                B_in = global_index(U.ptr, U.pitch, 0, ny - 1, border);
		for (int k = tid; k < nx+border*2; k+=threads){
			kk = k-border;
			kin = min(kk,nx-1);
			kin = max(kin,0);			
			B_out[kk] = B_in[kin];
		}

        }

	// WEST
	for (int k = tid; k < ny+border*2; k+= threads){
		kk = k-border;
        	B_out = global_index(U.ptr, U.pitch, 0, kk, border); 	
		kin = min(kk,nx-1);
		kin = max(kin,0);			
		for (int b = 0; b < border; b++)
                	B_out[-1-b] = global_index(U.ptr, U.pitch, 0, kin, border)[0];                      
        }

	// EAST
        for (int k = tid; k < ny+border*2; k+= threads){
		kk = k-border;
                B_out = global_index(U.ptr, U.pitch, nx, kk, border);     
		kin = min(kk,nx-1);
		kin = max(kin,0);			
                for (int b = 0; b < border; b++)
                        B_out[b] = global_index(U.ptr, U.pitch, nx - 1, kin,border)[0];
        }
}

void callSetBCOpen(dim3 grid, dim3 block, gpu_raw_ptr U, unsigned int NX, unsigned int NY, int border){
	setBCOpen<<<grid, block>>>(U, NX, NY, border);
}	 


void callCollectiveSetBCOpen(dim3 grid, dim3 block, const collBCKernelArgs* arg){

	//hipMemcpyToSymbolAsync(HIP_SYMBOL(bc_ctx), arg->, sizeof(collBCKernelArgs), 0, hipMemcpyHostToDevice);	
	callSetBCOpen(grid, block, arg->U0, arg->NX, arg->NY, arg->global_border); 
	callSetBCOpen(grid, block, arg->U1, arg->NX, arg->NY, arg->global_border);
        callSetBCOpen(grid, block, arg->U2, arg->NX, arg->NY, arg->global_border);
        callSetBCOpen(grid, block, arg->U3, arg->NX, arg->NY, arg->global_border);
}


// Set wall boundry condition
__global__ void setBCWall(gpu_raw_ptr U, unsigned int NX, unsigned int NY, int border){

	int threads = blockDim.x*blockDim.y;	

	float* B_in;
	float* B_out;

	int nx = NX-2*border;
	int ny = NY-2*border;

	int tid = threadIdx.y*blockDim.x+threadIdx.x;

	int kin;

	// SOUTH
	for (int b = 0; b < border; b++){
		B_out = global_index(U.ptr, U.pitch, 0, -1 - b, border);   
		B_in = global_index(U.ptr, U.pitch, 0, 0 + b, border);
		for (int k = tid-2; k < nx+border; k+=threads){
			kin = min(k,nx-1);
			kin = max(kin,0);			
			B_out[k] = B_in[kin];
		}
	}
	// NORTH
	for (int b = 0; b < border; b++){
                B_out = global_index(U.ptr, U.pitch, 0, ny + b, border);   
                B_in = global_index(U.ptr, U.pitch, 0, ny - 1 - b, border);
		for (int k = tid-2; k < nx+border; k+=threads){
			kin = min(k,nx-1);
			kin = max(kin,0);			
			B_out[k] = B_in[kin];
		}

        }

	// WEST
	for (int k = tid-2; k < ny; k+= threads){
		printf("k: %i", k);
        	B_out = global_index(U.ptr, U.pitch, 0, k, border); 	
		kin = min(k,nx-1);
		kin = max(kin,0);			
		for (int b = 0; b < border; b++)
                	B_out[-1-b] = global_index(U.ptr, U.pitch, 0 + b, kin, border)[0];                      
        }

	// EAST
        for (unsigned int k = tid; k < ny; k+= threads){
                B_out = global_index(U.ptr, U.pitch, nx, k, border);     
		kin = min(k,nx-1);
		kin = max(kin,0);			
                for (int b = 0; b < border; b++)
                        B_out[b] = global_index(U.ptr, U.pitch, nx - 1 - b, kin,border)[0];
        }

}

void callSetBCWall(dim3 grid, dim3 block, gpu_raw_ptr U, unsigned int NX, unsigned int NY, int border){
	setBCWall<<<grid, block>>>(U, NX, NY, border);
}	 


void callCollectiveSetBCWall(dim3 grid, dim3 block, const collBCKernelArgs* arg){

	callSetBCWall(grid, block, arg->U0, arg->NX, arg->NY, arg->global_border); 
	callSetBCWall(grid, block, arg->U1, arg->NX, arg->NY, arg->global_border);
        callSetBCWall(grid, block, arg->U2, arg->NX, arg->NY, arg->global_border);
        callSetBCWall(grid, block, arg->U3, arg->NX, arg->NY, arg->global_border);
}
